#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "optimizer.h"
#include "types.h"
#include "cuda_helper.h"

LegionRuntime::Logger::Category log_parameter("optimizer");

__global__
void add_kernel(int count, DATATYPE scale,
                const DATATYPE* src,
                DATATYPE* dst)
{
  CUDA_KERNEL_LOOP(i, count)
  {
    dst[i] += src[i] * scale;
  }
}

__global__
void scale_kernel(int count, DATATYPE a, DATATYPE b,
                  DATATYPE* ptr)
{
  CUDA_KERNEL_LOOP(i, count)
  {
    ptr[i] = (b - a) * ptr[i] + a;
  }
}

__global__
void adam_update(int count, DATATYPE alpha_t,
                 DATATYPE beta1, DATATYPE beta2, DATATYPE epsilon,
                 const DATATYPE *WGrad, DATATYPE *M,
                 DATATYPE *V, DATATYPE *W)
{
  CUDA_KERNEL_LOOP(i, count)
  {
    DATATYPE gt = WGrad[i];
    DATATYPE mt = beta1 * M[i] + (1 - beta1) * gt;
    DATATYPE vt = beta2 * V[i] + (1 - beta2) * gt * gt;
    M[i] = mt;
    V[i] = vt;
    W[i] -= alpha_t * mt / (sqrt(vt) + epsilon);
  }
}

__host__
void AdamOptimizer::update_task(const Task* task,
                                const std::vector<PhysicalRegion>& regions,
                                Context ctx, Runtime* runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const AdamOptimizer* op = (AdamOptimizer*) task->args;
  TensorAccessorRO<DATATYPE, 1> accWGrad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, NULL);
  TensorAccessorRW<DATATYPE, 1> accW(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, NULL);
  TensorAccessorRW<DATATYPE, 1> accV(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, NULL);
  TensorAccessorRW<DATATYPE, 1> accM(
      regions[3], task->regions[3], FID_DATA, ctx, runtime, NULL);
  int numReplicas = accWGrad.rect.volume() / accW.rect.volume();
  // Step 1: gather gradients in the first replica
  for (int i = 1; i < numReplicas; i++) {
    const DATATYPE* src = accWGrad.ptr + i * accW.rect.volume();
    add_kernel<<<GET_BLOCKS(accW.rect.volume()), CUDA_NUM_THREADS>>>(
        accW.rect.volume(), 1.0f, src, (DATATYPE*)accWGrad.ptr);
  }
  adam_update<<<GET_BLOCKS(accW.rect.volume()), CUDA_NUM_THREADS>>>(
      accW.rect.volume(), op->alpha_t, op->beta1, op->beta2, op->epsilon,
      accWGrad.ptr, accM.ptr, accV.ptr, accW.ptr);
}


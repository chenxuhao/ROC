/* Copyright 2019 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gnn.h"
#include "cuda_helper.h"

__host__
void Activation::forward_task(const Task *task,
                              const std::vector<PhysicalRegion>& regions,
                              Context ctx, Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Activation* op = (Activation*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  manager->reset();
  TensorAccessorR<DATATYPE, 2> accInput(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorW<DATATYPE, 2> accOutput(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager,
      false/*readOutput*/);
  assert(accInput.memory.kind() == Memory::Z_COPY_MEM);
  assert(accOutput.memory.kind() == Memory::Z_COPY_MEM);
  assert(accInput.rect == accOutput.rect);
  V_ID rowLeft = accInput.rect.lo[1], rowRight = accInput.rect.hi[1];
  int hiddenDim = accInput.rect.hi[0] - accInput.rect.lo[0] + 1;
  double ts_start = Realm::Clock::current_time_in_microseconds();
  hipdnnTensorDescriptor_t inTensor;
  hipdnnActivationDescriptor_t actiDesc;
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnCreateTensorDescriptor(&inTensor));
  int dims[] = {(int)(rowRight - rowLeft + 1), hiddenDim, 1};
  int strides[] = {dims[1] * dims[2], dims[2], 1};
  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnSetTensorNdDescriptor(inTensor, HIPDNN_DATA_FLOAT,
      3, dims, strides));
  switch (op->actiMode) {
    case AC_MODE_RELU:
      checkCUDNN(hipdnnSetActivationDescriptor(
          actiDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
      break;
    case AC_MODE_SIGMOID:
      checkCUDNN(hipdnnSetActivationDescriptor(
          actiDesc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));
      break;
    default:
      assert(false);
  }
  double ts_end = Realm::Clock::current_time_in_microseconds();
  printf("[Activation:forward] preprocess(%.2lfus)\n", ts_end - ts_start);
  checkCUDNN(hipdnnActivationForward(manager->dnn, actiDesc,
                                    &alpha, inTensor, accInput.fbCache,
                                    &beta, inTensor, accOutput.fbCache));
  checkCUDA(hipMemcpy(accOutput.ptr, accOutput.fbCache,
                       accOutput.rect.volume() * sizeof(DATATYPE),
                       hipMemcpyDeviceToHost));
  checkCUDNN(hipdnnDestroyTensorDescriptor(inTensor));
  checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
}

__host__
void Activation::backward_task(const Task *task,
                               const std::vector<PhysicalRegion>& regions,
                               Context ctx, Runtime* runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Activation* op = (Activation*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  manager->reset();
  TensorAccessorR<DATATYPE, 2> accOutputGrad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorR<DATATYPE, 2> accOutput(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager);
  TensorAccessorR<DATATYPE, 2> accInput(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, manager);
  TensorAccessorW<DATATYPE, 2> accInputGrad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime, manager,
      !(op->resetInputGrads[0])/*readOutput*/);
  assert(accOutput.memory.kind() == Memory::Z_COPY_MEM);
  assert(accOutputGrad.memory.kind() == Memory::Z_COPY_MEM);
  assert(accInput.memory.kind() == Memory::Z_COPY_MEM);
  assert(accInputGrad.memory.kind() == Memory::Z_COPY_MEM);
  assert(accOutput.rect == accOutputGrad.rect);
  assert(accOutput.rect == accInput.rect);
  assert(accOutput.rect == accInputGrad.rect);
  double ts_start = Realm::Clock::current_time_in_microseconds();
  V_ID rowLeft = accOutput.rect.lo[1], rowRight = accOutput.rect.hi[1];
  int hiddenDim = accOutput.rect.hi[0] - accOutput.rect.lo[0] + 1;
  float alpha = 1.0f, beta = 0.0f;
  hipdnnTensorDescriptor_t outTensor;
  hipdnnActivationDescriptor_t actiDesc;
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outTensor));
  int dims[] = {(int)(rowRight - rowLeft + 1), hiddenDim, 1};
  int strides[] = {dims[1] * dims[2], dims[2], 1};
  checkCUDNN(hipdnnSetTensorNdDescriptor(outTensor, HIPDNN_DATA_FLOAT,
        3, dims, strides));
  switch (op->actiMode) {
    case AC_MODE_RELU:
      checkCUDNN(hipdnnSetActivationDescriptor(
          actiDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
      break;
    case AC_MODE_SIGMOID:
      checkCUDNN(hipdnnSetActivationDescriptor(
          actiDesc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));
      break;
    default:
      assert(false);
  }
  double ts_end = Realm::Clock::current_time_in_microseconds();
  printf("[Activation:backward] preprocess(%.2lfus)\n", ts_end - ts_start);
  checkCUDNN(hipdnnActivationBackward(manager->dnn, actiDesc,
      &alpha, outTensor, accOutput.fbCache,
      outTensor, accOutputGrad.fbCache,
      outTensor, accInput.fbCache,
      &alpha, outTensor, accInputGrad.fbCache));
  checkCUDA(hipMemcpy(accInputGrad.ptr, accInputGrad.fbCache,
                       accInputGrad.rect.volume() * sizeof(DATATYPE),
                       hipMemcpyDeviceToHost));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outTensor));
  checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
  for (int i = 0; i < 8; i++)
    for (int j = 0; j < 8; j++)
      printf("[Activation:backward](%d, %d): outputGrad(%.4lf) output(%.4lf) input(%.4lf) inputGrad(%.4lf)\n",
             i, j, accOutputGrad.ptr[i*hiddenDim+j], accOutput.ptr[i*hiddenDim+j],
             accInput.ptr[i*hiddenDim+j], accInputGrad.ptr[i*hiddenDim+j]);
}

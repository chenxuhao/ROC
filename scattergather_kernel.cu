#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gnn.h"
#include "cuda_helper.h"
#include <hipcub/hipcub.hpp>

__global__
void copy_kernel(DATATYPE* dst,
                 const DATATYPE* src,
                 size_t size)
{
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
       i+= blockDim.x * gridDim.x)
  {
    dst[i] = src[i];
  }
}

__global__
void aggre_coop_kernel(V_ID rowLeft,
                       V_ID rowRight,
                       E_ID colLeft,
                       int hiddenDim,
                       const NodeStruct* row_ptrs,
                       const EdgeStruct* col_idxs,
                       const DATATYPE* input,
                       DATATYPE* output)
{
  assert(blockDim.x % hiddenDim == 0);
  //assert(aggrType == AGGR_SUM || aggrType == AGGR_AVG);
  int vtxPerBlock = blockDim.x / hiddenDim;
  typedef hipcub::BlockScan<E_ID, CUDA_NUM_THREADS> BlockScan;
  __shared__ BlockScan::TempStorage temp_storage;
  __shared__ E_ID blkColStart;
  __shared__ DATATYPE acc_h[CUDA_NUM_THREADS];
  int tidDiv = threadIdx.x / hiddenDim;
  int tidMod = threadIdx.x % hiddenDim;
  for (V_ID blkRowStart = blockIdx.x * vtxPerBlock + rowLeft;
       blkRowStart <= rowRight;
       blkRowStart += vtxPerBlock * gridDim.x)
  {
    E_ID myNumEdges = 0, scratchOffset, totalNumEdges = 0;
    if (threadIdx.x + blkRowStart <= rowRight && threadIdx.x < vtxPerBlock) {
      V_ID curVtx = threadIdx.x + blkRowStart;
      E_ID startColIdx, endColIdx = row_ptrs[curVtx-rowLeft].index;
      if (curVtx == rowLeft)
        startColIdx = colLeft;
      else
        startColIdx = row_ptrs[curVtx-rowLeft-1].index;
      myNumEdges = endColIdx - startColIdx;
      if (threadIdx.x == 0)
        blkColStart = startColIdx;
    }
    //if (myNumEdges > 0) printf("tid(%d) myNumEdges(%d)\n", threadIdx.x, myNumEdges);
    acc_h[threadIdx.x] = 0.0f;
    __syncthreads();
    BlockScan(temp_storage).ExclusiveSum(myNumEdges, scratchOffset, totalNumEdges);
    E_ID done = 0;
    while (totalNumEdges > 0) {
      if (tidDiv < totalNumEdges) {
        EdgeStruct es = col_idxs[blkColStart + done + tidDiv - colLeft];
        DATATYPE val = input[es.src * hiddenDim + tidMod];
        int offset = (es.dst - blkRowStart) * hiddenDim + tidMod;
        atomicAdd(&acc_h[offset], val);
      }
      done += vtxPerBlock;
      totalNumEdges -= (totalNumEdges > vtxPerBlock) ? vtxPerBlock : totalNumEdges;
    }
    __syncthreads();
    if (tidDiv + blkRowStart <= rowRight) {
      output[(blkRowStart-rowLeft)*hiddenDim+threadIdx.x] = acc_h[threadIdx.x];
    }
  }
}

__host__
void ScatterGather::forward_task(const Task *task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  //const ScatterGather* op = (ScatterGather*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  manager->reset();
  TensorAccessorRO<NodeStruct, 1> accRowPtr(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorRO<EdgeStruct, 1> accColIdx(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager);
  TensorAccessorRO<DATATYPE, 2> accInput(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, manager);
  TensorAccessorWO<DATATYPE, 2> accOutput(
      regions[3], task->regions[3], FID_DATA, ctx, runtime, manager);
  // Check memories are correctly mapped
  assert(accRowPtr.memory.kind() == Memory::GPU_FB_MEM);
  assert(accColIdx.memory.kind() == Memory::GPU_FB_MEM);
  assert(accInput.memory.kind() == Memory::Z_COPY_MEM);
  assert(accOutput.memory.kind() == Memory::Z_COPY_MEM);
#ifdef DEADCODE
  const AccessorRO<NodeStruct, 1> accRowPtr(regions[0], FID_DATA);
  const AccessorRO<EdgeStruct, 1> accColIdx(regions[1], FID_DATA);
  const AccessorRO<DATATYPE, 2> accInput(regions[2], FID_DATA);
  const AccessorWO<DATATYPE, 2> accOutput(regions[3], FID_DATA);
  Rect<1> rectRowPtr = runtime->get_index_space_domain(
                             ctx, task->regions[0].region.get_index_space());
  Rect<1> rectColIdx = runtime->get_index_space_domain(
                             ctx, task->regions[1].region.get_index_space());
  Rect<2> rectInput = runtime->get_index_space_domain(
                            ctx, task->regions[2].region.get_index_space());
  Rect<2> rectOutput = runtime->get_index_space_domain(
                            ctx, task->regions[3].region.get_index_space());
  assert(accRowPtr.accessor.is_dense_arbitrary(rectRowPtr));
  assert(accColIdx.accessor.is_dense_arbitrary(rectColIdx));
  assert(accInput.accessor.is_dense_arbitrary(rectInput));
  assert(accOutput.accessor.is_dense_arbitrary(rectOutput));
  const NodeStruct* rowPtrs = accRowPtr.ptr(rectRowPtr);
  const EdgeStruct* colIdxs = accColIdx.ptr(rectColIdx);
  const DATATYPE* zcInput = accInput.ptr(rectInput);
  DATATYPE* zcOutput = accOutput.ptr(rectOutput);
#endif
  V_ID rowLeft = accRowPtr.rect.lo[0], rowRight = accRowPtr.rect.hi[0];
  E_ID colLeft = accColIdx.rect.lo[0], colRight = accColIdx.rect.hi[0];
  int hiddenDim = accInput.rect.hi[0]-accInput.rect.lo[0]+1;
  assert(hiddenDim == accOutput.rect.hi[0]-accOutput.rect.lo[0]+1);
  assert(accOutput.rect.lo[1] == accRowPtr.rect.lo[0]);
  assert(accOutput.rect.hi[1] == accRowPtr.rect.hi[0]);

  copy_kernel<<<GET_BLOCKS(accInput.rect.volume()), CUDA_NUM_THREADS>>>(
      accInput.fbCache, accInput.ptr, accInput.rect.volume());
  aggre_coop_kernel<<<GET_BLOCKS(accOutput.rect.volume()), CUDA_NUM_THREADS>>>(
      rowLeft, rowRight, colLeft, hiddenDim, accRowPtr.ptr, accColIdx.ptr,
      accInput.fbCache, accOutput.fbCache);
  // Need to copy results back to new_pr
  hipDeviceSynchronize();
  checkCUDA(hipMemcpy(accOutput.ptr, accOutput.fbCache,
                       accOutput.rect.volume() * sizeof(DATATYPE),
                       hipMemcpyDeviceToHost));
  //copy_kernel<<<GET_BLOCKS(rectOutput.volume()), CUDA_NUM_THREADS>>>(
  //    zcOuptut, manager->fbCache[outputId], rectOutput.volume());
}

__host__
void ScatterGather::backward_task(const Task *task,
                                  const std::vector<PhysicalRegion> &regions,
                                  Context ctx, Runtime *runtime)
{
  // Forward and backward do exact same thing
  return forward_task(task, regions, ctx, runtime);
}

__host__
void ScatterGather::update_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
}                         

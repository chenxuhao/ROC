#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gnn.h"
#include "cuda_helper.h"

__global__
void softmax_backward(DATATYPE* logitsGrad,
                      const DATATYPE* labels,
                      const int* mask,
                      int hiddenDim,
                      V_ID numVertices)
{
  CUDA_KERNEL_LOOP(i, hiddenDim * numVertices)
  {
    logitsGrad[i] -= labels[i];
    int myVtxID = i % hiddenDim;
    if (mask[myVtxID] == 0)
      logitsGrad[i] = 0;
  }
}

__host__
void SoftmaxCrossEntropy::backward_task(const Task *task,
                                        const std::vector<PhysicalRegion>& regions,
                                        Context ctx, Runtime* runtime)
{
  assert(regions.size() == 3 or regions.size() == 4);
  assert(regions.size() == task->regions.size());
  const SoftmaxCrossEntropy* op = (SoftmaxCrossEntropy*) task->args;
  // assert the three inputs need reset gradient
  assert(op->resetInputGrads[0]);
  assert(op->resetInputGrads[1]);
  assert(op->resetInputGrads[2]);
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  manager->reset();
  TensorAccessorRO<DATATYPE, 2> accLogits(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorRO<DATATYPE, 2> accLabels(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager);
  TensorAccessorWO<DATATYPE, 2> accLogitsGrad(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, manager);
  assert(accLogits.memory.kind() == Memory::Z_COPY_MEM);
  assert(accLabels.memory.kind() == Memory::Z_COPY_MEM);
  assert(accLogitsGrad.memory.kind() == Memory::Z_COPY_MEM);
  V_ID rowLeft = accLogits.rect.lo[1], rowRight = accLogits.rect.hi[1];
  int hiddenDim = accLogits.rect.hi[0] - accLogits.rect.lo[0] + 1;
  if (regions.size() == 4) {
    TensorAccessorRO<int, 2> accMask(
        regions[3], task->regions[3], FID_DATA, ctx, runtime, manager);
    assert(accLogits.rect == accLabels.rect);
    assert(accLogits.rect == accLogitsGrad.rect);
    assert(accMask.rect.lo[0] == accMask.rect.hi[0]);
    assert(accMask.rect.lo[1] == rowLeft);
    assert(accMask.rect.hi[1] == rowRight);
    
    hipdnnTensorDescriptor_t inputDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    int dims[] = {(int)(rowRight - rowLeft + 1), hiddenDim, 1, 1};
    int strides[] = {dims[1] * dims[2] * dims[3], dims[2] * dims[3], dims[3], 1};
    checkCUDNN(hipdnnSetTensorNdDescriptor(inputDesc, HIPDNN_DATA_FLOAT,
        4, dims, strides));
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnSoftmaxForward(manager->dnn, HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, inputDesc, accLogits.fbCache,
        &beta, inputDesc, accLogitsGrad.fbCache));
    softmax_backward<<<GET_BLOCKS(accLogits.rect.volume()), CUDA_NUM_THREADS>>>(
        accLogitsGrad.fbCache, accLabels.fbCache, accMask.fbCache,
        hiddenDim, rowRight - rowLeft + 1);
  } else {
    assert(false);
  }
}

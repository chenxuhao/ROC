#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gnn.h"
#include "cuda_helper.h"

__global__
void op_kernel(const DATATYPE* input0,
               const DATATYPE* input1,
               DATATYPE* output,
               coord_t size,
               ElementType type)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    switch(type) {
      case EW_TYPE_ADD:
        output[i] = input0[i] + input1[i];
        break;
      case EW_TYPE_MUL:
        output[i] = input0[i] * input1[i];
        break;
      default:
        assert(false);
    }
  }
}

__host__
void Element::forward_task(const Task *task,
                           const std::vector<PhysicalRegion>& regions,
                           Context ctx, Runtime* runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Element* op = (Element*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  manager->reset();
  TensorAccessorR<DATATYPE, 2> accInput0(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorR<DATATYPE, 2> accInput1(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager);
  TensorAccessorW<DATATYPE, 2> accOutput(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, manager,
      false/*readOutput*/);
  assert(accInput0.rect == accInput1.rect);
  assert(accOutput.rect == accInput0.rect);
  assert(accInput0.memory.kind() == Memory::Z_COPY_MEM);
  assert(accInput1.memory.kind() == Memory::Z_COPY_MEM);
  assert(accOutput.memory.kind() == Memory::Z_COPY_MEM);
  op_kernel<<<GET_BLOCKS(accOutput.rect.volume()), CUDA_NUM_THREADS>>>(
      accInput0.fbCache, accInput1.fbCache, accOutput.fbCache,
      accOutput.rect.volume(), op->elementType);
  checkCUDA(hipMemcpy(accOutput.ptr, accOutput.fbCache,
                       accOutput.rect.volume() * sizeof(DATATYPE),
                       hipMemcpyDeviceToHost));
}

__host__
void Element::backward_task(const Task *task,
                            const std::vector<PhysicalRegion>& regions,
                            Context ctx, Runtime* runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Element* op = (Element*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  manager->reset();
  TensorAccessorR<DATATYPE, 2> accOutputGrad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorW<DATATYPE, 2> accInput0Grad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager,
      !(op->resetInputGrads[0])/*readOutput*/);
  TensorAccessorW<DATATYPE, 2> accInput1Grad(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, manager,
      !(op->resetInputGrads[1])/*readOutput*/);
  assert(accOutputGrad.rect == accInput0Grad.rect);
  assert(accOutputGrad.rect == accInput1Grad.rect);
  switch (op->elementType) {
    case EW_TYPE_ADD:
    {
      add_kernel<<<GET_BLOCKS(accOutputGrad.rect.volume()), CUDA_NUM_THREADS>>>(
          accInput0Grad.fbCache, accOutputGrad.fbCache, accOutputGrad.rect.volume());
      add_kernel<<<GET_BLOCKS(accOutputGrad.rect.volume()), CUDA_NUM_THREADS>>>(
          accInput1Grad.fbCache, accOutputGrad.fbCache, accOutputGrad.rect.volume());
      break;
    }
    case EW_TYPE_MUL:
    default:
      assert(false);
  }
  checkCUDA(hipMemcpy(accInput0Grad.ptr, accInput0Grad.fbCache,
                       accInput0Grad.rect.volume() * sizeof(DATATYPE),
                       hipMemcpyDeviceToHost));
  checkCUDA(hipMemcpy(accInput1Grad.ptr, accInput1Grad.fbCache,
                       accInput1Grad.rect.volume() * sizeof(DATATYPE),
                       hipMemcpyDeviceToHost));
}

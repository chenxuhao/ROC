#include "hip/hip_runtime.h"
#include "cuda_helper.h"
__global__
void scale_kernel(float* ptr, coord_t size, float a, float b)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    ptr[i] = (b - a) * ptr[i] + a;
  }
}

__global__
void assign_kernel(float* ptr, coord_t size, float value)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    ptr[i] = value;
  }
}


#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "initializer.h"
#include "types.h"
#include "cuda_helper.h"
#include <hiprand.h>
#include <ctime>

void GlorotUniform::init_task(const Task* task,
                              const std::vector<PhysicalRegion>& regions,
                              Context ctx, Runtime* runtime)
{
  assert(regions.size() == 1);
  assert(task->regions.size() == 1);
  TensorAccessorW<DATATYPE, 2> accW(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, NULL,
      false/*readOutput*/);
  int inputDim = accW.rect.hi[0] - accW.rect.lo[0] + 1;
  int outputDim = accW.rect.hi[1] - accW.rect.lo[1] + 1;
  // TODO: remove me
  //assign_kernel<<<GET_BLOCKS(accW.rect.volume()), CUDA_NUM_THREADS>>>(
  //    accW.ptr, accW.rect.volume(), 1.0/64);
  //return;
  //float scale = *((float*) task->args);
  float scale = sqrt(6.0 / (inputDim + outputDim));
  printf("scale = %.4lf\n", scale);
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  // TODO: change to random seed before releasing
  int seed = *((int*) task->args);
  fprintf(stderr, "seed = %d\n", seed);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  checkCUDA(hiprandGenerateUniform(gen, accW.ptr, accW.rect.volume()));
  scale_kernel<<<GET_BLOCKS(accW.rect.volume()), CUDA_NUM_THREADS>>>(
      accW.ptr, accW.rect.volume(), -scale, scale);
  checkCUDA(hipDeviceSynchronize());
  hiprandDestroyGenerator(gen);
}

void ZerosInitializer::init_task(const Task* task,
                                 const std::vector<PhysicalRegion>& regions,
                                 Context ctx, Runtime* runtime)
{
  assert(regions.size() == 1);
  assert(task->regions.size() == 1);
  TensorAccessorW<DATATYPE, 2> accW(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, NULL,
      false/*readOutput*/);
  assign_kernel<<<GET_BLOCKS(accW.rect.volume()), CUDA_NUM_THREADS>>>(
      accW.ptr, accW.rect.volume(), 0);
  checkCUDA(hipDeviceSynchronize());
}

void zero_grad_task_impl(const Task* task,
                         const std::vector<PhysicalRegion>& regions,
                         Context ctx, Runtime* runtime)
{
  assert(regions.size() == task->regions.size());
  for (size_t i = 0; i < regions.size(); i++) {
    Domain domain = runtime->get_index_space_domain(
        ctx, task->regions[i].region.get_index_space());
    DATATYPE* w;
    switch (domain.get_dim()) {
      case 0:
      {
        // Do not support 0-dim parameters
        assert(false);
        break;
      }
      case 1:
      {
        TensorAccessorW<DATATYPE, 1> accW(
            regions[i], task->regions[i], FID_DATA, ctx, runtime, NULL,
            false/*readOutput*/);
        w = accW.ptr;
        break;
      }
      case 2:
      {
        TensorAccessorW<DATATYPE, 2> accW(
            regions[i], task->regions[i], FID_DATA, ctx, runtime, NULL,
            false/*readOutput*/);
        w = accW.ptr;
        break;
      }
      case 3:
      {
        TensorAccessorW<DATATYPE, 3> accW(
            regions[i], task->regions[i], FID_DATA, ctx, runtime, NULL,
            false/*readOutput*/);
        w = accW.ptr;
        break;
      }
      default:
      {
         assert(false);
         break;
      }
    }
    assign_kernel<<<GET_BLOCKS(domain.get_volume()), CUDA_NUM_THREADS>>>(
        w, domain.get_volume(), 0.0f);
  }
  checkCUDA(hipDeviceSynchronize());
}

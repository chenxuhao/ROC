/* Copyright 2019 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gnn.h"
#include "cuda_helper.h"

__host__
void Dropout::init_task(const Task *task,
                        const std::vector<PhysicalRegion>& regions,
                        Context ctx, Runtime* runtime)
{
  assert(regions.size() == 1);
  assert(task->regions.size() == 1);
  const Dropout* op = (Dropout*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  manager->reset();
  TensorAccessorRO<DATATYPE, 2> accInput(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  assert(accInput.memory.kind() == Memory::Z_COPY_MEM);
  V_ID rowLeft = accInput.rect.lo[1], rowRight = accInput.rect.hi[1];
  int inDim = accInput.rect.hi[0] - accInput.rect.lo[0] + 1;
  hipdnnDropoutDescriptor_t dropoutDesc;
  hipdnnTensorDescriptor_t inputDesc;
  checkCUDNN(hipdnnCreateDropoutDescriptor(&dropoutDesc));
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
  checkCUDNN(hipdnnSetDropoutDescriptor(dropoutDesc, manager->dnn, op->rate,
      manager->dropoutStates, manager->dropoutSize, op->seed));
  int dims[] = {(int)(rowRight - rowLeft + 1), inDim, 1};
  int strides[] = {dims[1] * dims[2], dims[2], 1};
  checkCUDNN(hipdnnSetTensorNdDescriptor(inputDesc, HIPDNN_DATA_FLOAT,
      3, dims, strides));
  ResourceManager::ReservedSpace space;
  checkCUDNN(cudnnDropoutGetReserveSpaceSize(inputDesc, &(space.size)));
  off_t offset = manager->allocator->alloc_bytes(space.size);
  assert(offset >= 0);
  space.ptr = manager->allocator->get_direct_ptr(offset, 0);
  LogicalRegion my_lr = regions[0].get_logical_region();
  assert(manager->reservedSpace.find(my_lr) == manager->reservedSpace.end());
  manager->reservedSpace[my_lr] = space;
  manager->dropoutDesc[my_lr] = dropoutDesc;
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputDesc));
}

__host__
void Dropout::forward_task(const Task *task,
                           const std::vector<PhysicalRegion>& regions,
                           Context ctx, Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Dropout* op = (Dropout*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  TensorAccessorRO<DATATYPE, 2> accInput(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorWO<DATATYPE, 2> accOutput(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager);
  assert(accInput.memory.kind() == Memory::Z_COPY_MEM);
  assert(accOutput.memory.kind() == Memory::Z_COPY_MEM);
  V_ID rowLeft = accInput.rect.lo[1], rowRight = accInput.rect.hi[1];
  int inDim = accInput.rect.hi[0] - accInput.rect.lo[0] + 1;

  double ts_start = Realm::Clock::current_time_in_microseconds();
  hipdnnTensorDescriptor_t inputDesc, outputDesc;
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
  int dims[] = {(int)(rowRight - rowLeft + 1), inDim, 1};
  int strides[] = {dims[1] * dims[2], dims[2], 1};
  checkCUDNN(hipdnnSetTensorNdDescriptor(inputDesc, HIPDNN_DATA_FLOAT,
      3, dims, strides));
  checkCUDNN(hipdnnSetTensorNdDescriptor(outputDesc, HIPDNN_DATA_FLOAT,
      3, dims, strides));
  LogicalRegion my_lr = regions[0].get_logical_region();
  assert(manager->reservedSpace.find(my_lr) != manager->reservedSpace.end());
  ResourceManager::ReservedSpace space = manager->reservedSpace[my_lr];
  hipdnnDropoutDescriptor_t dropoutDesc = manager->dropoutDesc[my_lr];
  double ts_end = Realm::Clock::current_time_in_microseconds();
  printf("[Dropout::Fwd] pre-process %.4lfus\\n", ts_end - ts_start);
  checkCUDNN(cudnnDropoutForward(manager->dnn, dropoutDesc, inputDesc,
      accInput.fbCache, outputDesc, accOutput.fbCache, space.ptr, space.size));
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputDesc));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputDesc));
}

__host__
void Dropout::backward_task(const Task *task,
                            const std::vector<PhysicalRegion>& regions,
                            Context ctx, Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Dropout* op = (Dropout*) task->args;
  ResourceManager* manager = *((ResourceManager**) task->local_args);
  assert(manager->proc_id == task->current_proc.id);
  TensorAccessorRO<DATATYPE, 2> accOutputGrad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime, manager);
  TensorAccessorWO<DATATYPE, 2> accInputGrad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, manager);
  assert(accOutputGrad.memory.kind() == Memory::Z_COPY_MEM);
  assert(accInputGrad.memory.kind() == Memory::Z_COPY_MEM);
  V_ID rowLeft = accInputGrad.rect.lo[1], rowRight = accInputGrad.rect.hi[1];
  int inDim = accInputGrad.rect.hi[0] - accInputGrad.rect.lo[0] + 1;

  double ts_start = Realm::Clock::current_time_in_microseconds();
  hipdnnTensorDescriptor_t inputDesc, outputDesc;
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
  int dims[] = {(int)(rowRight - rowLeft + 1), inDim, 1};
  int strides[] = {dims[1] * dims[2], dims[2], 1};
  checkCUDNN(hipdnnSetTensorNdDescriptor(inputDesc, HIPDNN_DATA_FLOAT,
      3, dims, strides));
  checkCUDNN(hipdnnSetTensorNdDescriptor(outputDesc, HIPDNN_DATA_FLOAT,
      3, dims, strides));
  LogicalRegion my_lr = regions[0].get_logical_region();
  assert(manager->reservedSpace.find(my_lr) != manager->reservedSpace.end());
  ResourceManager::ReservedSpace space = manager->reservedSpace[my_lr];
  hipdnnDropoutDescriptor_t dropoutDesc = manager->dropoutDesc[my_lr];
  double ts_end = Realm::Clock::current_time_in_microseconds();
  printf("[Dropout::Bwd] pre-process %.4lfus\\n", ts_end - ts_start);
  checkCUDNN(cudnnDropoutBackward(manager->dnn, dropoutDesc, inputDesc,
      accOutputGrad.fbCache, outputDesc, accInputGrad.fbCache, space.ptr, space.size));
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputDesc));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputDesc));
}
